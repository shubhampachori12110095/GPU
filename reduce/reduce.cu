
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

// Interleaved addressing with divergent branching
__global__ void reduce_kernel0(int *d_out, int *d_in)
{
    extern __shared__ int s_data[];

    // thread ID inside the block
    unsigned int tid = threadIdx.x;
    // global ID across all blocks
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Copy elements from global memoery into per-block shared memory
    s_data[tid] = d_in[gid];
    // Ensure all elements have been copied into shared memory
    __syncthreads();

    // s = 1, 2, 4, 8, ..... blockDim.x / 2
    for (unsigned int s = 1; s < blockDim.x; s <<= 1) {
        if (tid % (s << 1) == 0) {
            s_data[tid] += s_data[tid + s];
        }
        // Ensure all threads in the block finish add in this round
        __syncthreads();
    }

    // write the reduction sum back to the global memory
    if (tid == 0) {
        d_out[blockIdx.x] = s_data[0];
    }
}

// Interleaved addressing with bank conflicts
__global__ void reduce_kernel1(int *d_out, int *d_in)
{
    extern __shared__ int s_data[];

    // thread ID inside the block
    unsigned int tid = threadIdx.x;
    // global ID across all blocks
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Copy elements from global memoery into per-block shared memory
    s_data[tid] = d_in[gid];
    // Ensure all elements have been copied into shared memory
    __syncthreads();    

    // s = 1, 2, 4, 8, ..... blockDim.x / 2
    for (unsigned int s = 1; s < blockDim.x; s <<= 1) {
        int index = tid * s * 2;

        if (index + s < blockDim.x) {
            s_data[index] += s_data[index + s];
        }

        // Ensure all threads in the block finish add in this round
        __syncthreads();
    }

    if (tid == 0) {
        d_out[blockIdx.x] = s_data[0];
    }
}

// Sequential addressing
__global__ void reduce_kernel2(int *d_out, int *d_in)
{
    extern __shared__ int s_data[];

    // thread ID inside the block
    unsigned int tid = threadIdx.x;
    // global ID across all blocks
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Copy elements from global memoery into per-block shared memory
    s_data[tid] = d_in[gid];
    // Ensure all elements have been copied into shared memory
    __syncthreads();    

    // s = blockDim.x / 2, ....., 8, 4, 2, 1
    for (unsigned int s = (blockDim.x >> 1); s >= 1; s >>= 1) {
        if (tid < s) {
            s_data[tid] += s_data[tid + s];
        }
        // Ensure all threads in the block finish add in this round
        __syncthreads();
    }

    if (tid == 0) {
        d_out[blockIdx.x] = s_data[0];
    }
}

// First add during global load
__global__ void reduce_kernel3(int *d_out, int *d_in)
{
    extern __shared__ int s_data[];

    // thread ID inside the block
    unsigned int tid = threadIdx.x;
    // global ID across all blocks
    unsigned int gid = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    s_data[tid] = d_in[gid] + d_in[gid + blockDim.x];
    // Ensure all elements have been copied into shared memory
    __syncthreads();    

    // s = blockDim.x / 2, ....., 8, 4, 2, 1
    for (unsigned int s = (blockDim.x >> 1); s >= 1; s >>= 1) {
        if (tid < s) {
            s_data[tid] += s_data[tid + s];
        }
        // Ensure all threads in the block finish add in this round
        __syncthreads();
    }

    if (tid == 0) {
        d_out[blockIdx.x] = s_data[0];
    }    
}

inline bool is_power_of_2(int n)
{
    return ((n & (n - 1)) == 0);
}


// input: array (in host memory), array size, expected result, kernel function ID and iterations 
void reduce(int *h_in, int array_size, int expected_result, int kernel_id, int iters)
{
    // # of threads per block. It should be the power of two
    int threads = 1 << 10;
    // # of blocks in total. 
    int blocks = 1;
    // GPU memory pointers
    int *d_in, *d_intermediate, *d_out;
    // final result in host memory
    int h_out;

    if (!h_in || array_size <= 0 || !is_power_of_2(array_size))
        goto out;

    if (array_size > threads)
        blocks = array_size / threads;
    
    // allocate GPU memory
    if (hipMalloc((void**) &d_in, array_size * sizeof(int)) != hipSuccess
     || hipMalloc((void**) &d_intermediate, blocks * sizeof(int)) != hipSuccess
     || hipMalloc((void**) &d_out, sizeof(int)) != hipSuccess)
        goto out;
    

    // copy the input array from the host memory to the GPU memory
    hipMemcpy(d_in, h_in, array_size * sizeof(int), hipMemcpyHostToDevice);

    // run many times
    for (int i = 0; i < iters; i++) {
        switch (kernel_id) {
            // Interleaved addressing with divergent branching 
            case 0: 
                // first stage reduce
                reduce_kernel0<<<blocks, threads, threads * sizeof(int)>>>(d_intermediate, d_in);
                // second stage reduce    
                reduce_kernel0<<<1, blocks, blocks * sizeof(int)>>>(d_out, d_intermediate);
                break;
            // Interleaved addressing with bank conflicts
            case 1:
                reduce_kernel1<<<blocks, threads, threads * sizeof(int)>>>(d_intermediate, d_in);
                reduce_kernel1<<<1, blocks, blocks * sizeof(int)>>>(d_out, d_intermediate);
                break;  
            // Sequential addressing              
            case 2:
                reduce_kernel2<<<blocks, threads, threads * sizeof(int)>>>(d_intermediate, d_in);
                reduce_kernel2<<<1, blocks, blocks * sizeof(int)>>>(d_out, d_intermediate);
                break;
            // First add during global load
            case 3:
                reduce_kernel3<<<blocks, threads / 2 , threads / 2 * sizeof(int)>>>(d_intermediate, d_in);
                reduce_kernel3<<<1, blocks / 2, blocks / 2 * sizeof(int)>>>(d_out, d_intermediate);  
                break;              
            default:
                printf("Invalid kernel function ID %d\n", kernel_id);   
                goto out;      
        }
    }

    // copy the result from the GPU memory to the host memory
    hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);

    if (h_out != expected_result) {
        printf("Wrong result: %d (expected) %d (actual)\n", expected_result, h_out);
    }

out:
    // free GPU memory
    hipFree(d_in);
    hipFree(d_intermediate);
    hipFree(d_out);
}

// generate a random integer in [min, max]
inline int random_range(int min, int max)
{
    if (min > max)
        return 0;
    else
        return min + rand() / (RAND_MAX / (max - min + 1) + 1);
}

int main(int argc, char **argv) 
{
    if (argc != 3) {
        printf("%s [kernel ID] [iterations]\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    int kernel_id = atoi(argv[1]);
    int iters = atoi(argv[2]);
    if (iters <= 0 || kernel_id < 0) {
        printf("Invalid input\n");
        exit(EXIT_FAILURE);
    }

    const int ARRAY_SIZE = 1 << 20;
    int h_in[ARRAY_SIZE];
    int sum = 0;
    
    // initialize random number generator
    srand(time(NULL));
    int min = 0, max = 10;

    for (int i = 0; i < ARRAY_SIZE; i++) {
        // generate a random int in a range
        h_in[i] = random_range(min, max);
        sum += h_in[i];
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);
    reduce(h_in, ARRAY_SIZE, sum, kernel_id, iters);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float elapsed_time;
    hipEventElapsedTime(&elapsed_time, start, stop);    
    elapsed_time /= iters;      

    printf("Average time elapsed: %f ms\n", elapsed_time);

    return 0;
}